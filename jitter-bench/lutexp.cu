#include "hip/hip_runtime.h"
#include <stdio.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include "lutexp.h"
#include "lutexp.cu.h"

#define MAX_BLOCKS 256
#define THREADS_PER_BLOCK 256
#define WARP_SIZE 32
#define NUM_WARPS THREADS_PER_BLOCK/WARP_SIZE

#define CUDA_RC(rc) if( (rc) != hipSuccess ) \
  {fprintf(stderr, "Error %s at %s line %d\n", hipGetErrorString(hipGetLastError()), __FILE__,__LINE__); MPI_Abort(MPI_COMM_WORLD, 1);}


/*=============================================================*/
/* Algorithm for exp(x):                                       */
/*                                                             */
/*       x*recln2 = p + r                                      */
/*       exp(x) = (2**p) * exp(r*ln2)                          */
/*                                                             */
/*       where p = integer + n-bit fraction                    */
/*       and r is a small remainder.                           */
/*                                                             */
/*       The integer part of p goes in the exponent field,     */
/*       the n-bit fraction is handled with a lookup table.    */
/*       The remainder term gets a Taylor expansion.           */
/*                                                             */
/* Use an 8-bit fraction, and a 5th order Taylor series.       */
/*                                                             */
/* p = (2**44 + 2**43) + (x*recln2) - (2**44 + 2**43)          */
/* gives 8-bit accuracy in the n-bit fraction                  */
/*=============================================================*/

__device__ double lutexp(double x, double * table)
{
   int hind;
   double result, hfactor;
   double recln2 = 1.44269504088896340736;
   double twop44_plus_twop43 = 2.6388279066624000e+13;

   struct uPair { unsigned lo ; unsigned hi ; };

   union { double d; struct uPair up; } X;
   union { double d; struct uPair up; } Result;

   double t, poly;
   double f0, f1, f2, f3, f4;
   double c2 = 1.0/2.0;
   double c3 = 1.0/6.0;
   double c4 = 1.0/24.0;
   double c5 = 1.0/120.0;

   /*--------------------------------------------------------*/
   /* multiply the input value by the reciprocal of ln(2)    */
   /* and shift by 2**44 + 2**43; save the exponent          */
   /*--------------------------------------------------------*/
   X.d = x*recln2 + twop44_plus_twop43;
   Result.up.hi = ( ( (X.up.lo >> 8) + 1023 )  << 20 ) & 0x7ff00000;
   Result.up.lo = 0;

   /*--------------------------------------------------*/
   /* compute the small remainder for the polynomial   */
   /* use the last 8 bits of the shifted X as an index */
   /*--------------------------------------------------*/
   t = x - (X.d - twop44_plus_twop43)*M_LN2;
   hind = X.up.lo & 0x000000ff;
   hfactor = Result.d * table[hind];

// /*---------------------------------------*/
// /* use a polynomial expansion for exp(t) */
// /*---------------------------------------*/
// poly = 1.0 + t*(c1 + t*(c2 + t*(c3 + t*(c4 + t*c5))));

   /*----------------------------------------------------*/
   /* for a single call, better to factor the polynomial */
   /*----------------------------------------------------*/
   f0 = 1.0 + t;
   f2 = t*t;
   f4 = f2*f2;
   f1 = c2 + t*c3;
   f3 = c4 + t*c5;

   poly = (f0 + f1*f2) + f3*f4;

   /*---------------------------------*/
   /* construct the result and return */
   /*---------------------------------*/
   result = hfactor* poly;

   /*-----------------------------------*/
   /* check input value for valid range */
   /*-----------------------------------*/
   if      (x < -709.0) return 0.0;
   else if (x >  709.0) return HUGE_VAL;
   else return result;
}



// ----------------------------------------------------------
// routine for computation; data is returned in the out array
// which must be copied back to the host for final reduction
// ----------------------------------------------------------
__global__ void compute(long npts, int nrand, double * xrand, double * table)
{
  double tsum = 0.0;
  long lrand, ndx;
  lrand = (long) nrand;
  for (long i = blockDim.x * blockIdx.x + threadIdx.x; i < npts; i += blockDim.x * gridDim.x) {
    ndx = i % lrand;
    tsum = tsum + lutexp(xrand[ndx], table);
  }
  static volatile double keep = 0.0;
  keep += tsum; // keep compiler from discarding loop
  __syncthreads();
}


double * block_data;
double * dev_xrand, * dev_table, * dev_block_data;
int nrand;
int numBlocks;

void __config_lutexp_cuda(double *xrand, int _nrand)
{
  nrand = _nrand;

  // each MPI rank selects a device
  int myrank;
  MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
  int numDevices, myDevice;
  CUDA_RC(hipGetDeviceCount(&numDevices));
  myDevice = myrank % numDevices;
  CUDA_RC(hipSetDevice(myDevice));

  double *table = (double *) malloc(sizeof(exp_table));

  // make a copy of the exp look-up table
  for (int i = 0; i < sizeof(exp_table) / sizeof(exp_table[0]); i++) table[i] = exp_table[i];

  // allocate GPU data
  CUDA_RC(hipMalloc((void **)&dev_table, sizeof(exp_table)));
  CUDA_RC(hipMalloc((void **)&dev_xrand, nrand*sizeof(double)));
  CUDA_RC(hipMalloc((void **)&dev_block_data, MAX_BLOCKS*sizeof(double)));

  block_data = (double *) malloc(MAX_BLOCKS*sizeof(double));

  // copy data to the GPU one time
  CUDA_RC(hipMemcpy(dev_table, table, sizeof(exp_table), hipMemcpyHostToDevice));
  CUDA_RC(hipMemcpy(dev_xrand, xrand, nrand*sizeof(double),  hipMemcpyHostToDevice));
}

void __compute_lutexp_cuda(uint64_t calib_iters /* == npts? */)
{
  int threadsPerBlock = THREADS_PER_BLOCK;
  int numBlocks = (calib_iters + threadsPerBlock - 1) / threadsPerBlock;
  if (numBlocks > MAX_BLOCKS) numBlocks = MAX_BLOCKS;
  compute<<<numBlocks, threadsPerBlock>>>(calib_iters, nrand, dev_xrand, dev_table);
  CUDA_RC(hipMemcpy(block_data, dev_block_data, numBlocks*sizeof(double), hipMemcpyDeviceToHost));
}

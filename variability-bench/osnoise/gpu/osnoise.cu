#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <mpi.h>
#include <pthread.h>
#include <unistd.h>
#include <sys/syscall.h>
#include <errno.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include "lutexp.h"

#define MAX_BLOCKS 256
#define THREADS_PER_BLOCK 256
#define WARP_SIZE 32
#define NUM_WARPS THREADS_PER_BLOCK/WARP_SIZE

#define BARRIER  10
#define EXCHANGE 11

#define SORT_ASCENDING_ORDER   1
#define SORT_DESCENDING_ORDER -1

#define CUDA_RC(rc) if( (rc) != hipSuccess ) \
  {fprintf(stderr, "Error %s at %s line %d\n", hipGetErrorString(hipGetLastError()), __FILE__,__LINE__); MPI_Abort(MPI_COMM_WORLD, 1);}

void get_local_communicator(int *, int *, MPI_Comm *);

__global__ void compute(long, int, double *, double *, double *);

__device__ double lutexp(double, double *);

void exch(int);

void print_help(void);

void sortx(double *, int, int *, int);

extern "C" void TMrand(unsigned long seed, int nrand, double * xrand);

static int myrank, myrow, mycol, npex, npey, exchcount;

static double * sendn,  * sende,  * sends,  * sendw,  * recvn,  * recve,  * recvs,  * recvw;


/*=============================================================*/
/* Algorithm for exp(x):                                       */
/*                                                             */
/*       x*recln2 = p + r                                      */
/*       exp(x) = (2**p) * exp(r*ln2)                          */
/*                                                             */
/*       where p = integer + n-bit fraction                    */
/*       and r is a small remainder.                           */
/*                                                             */
/*       The integer part of p goes in the exponent field,     */
/*       the n-bit fraction is handled with a lookup table.    */
/*       The remainder term gets a Taylor expansion.           */
/*                                                             */
/* Use an 8-bit fraction, and a 5th order Taylor series.       */
/*                                                             */
/* p = (2**44 + 2**43) + (x*recln2) - (2**44 + 2**43)          */
/* gives 8-bit accuracy in the n-bit fraction                  */
/*=============================================================*/

__device__ double lutexp(double x, double * table)
{
   int hind;
   double result, hfactor;
   double recln2 = 1.44269504088896340736;
   double twop44_plus_twop43 = 2.6388279066624000e+13;

   struct uPair { unsigned lo ; unsigned hi ; };

   union { double d; struct uPair up; } X;
   union { double d; struct uPair up; } Result;

   double t, poly;
   double f0, f1, f2, f3, f4;
   double c2 = 1.0/2.0;
   double c3 = 1.0/6.0;
   double c4 = 1.0/24.0;
   double c5 = 1.0/120.0;

   /*--------------------------------------------------------*/
   /* multiply the input value by the reciprocal of ln(2)    */
   /* and shift by 2**44 + 2**43; save the exponent          */
   /*--------------------------------------------------------*/
   X.d = x*recln2 + twop44_plus_twop43;
   Result.up.hi = ( ( (X.up.lo >> 8) + 1023 )  << 20 ) & 0x7ff00000;
   Result.up.lo = 0;

   /*--------------------------------------------------*/
   /* compute the small remainder for the polynomial   */
   /* use the last 8 bits of the shifted X as an index */
   /*--------------------------------------------------*/
   t = x - (X.d - twop44_plus_twop43)*M_LN2;
   hind = X.up.lo & 0x000000ff;
   hfactor = Result.d * table[hind];

// /*---------------------------------------*/
// /* use a polynomial expansion for exp(t) */
// /*---------------------------------------*/
// poly = 1.0 + t*(c1 + t*(c2 + t*(c3 + t*(c4 + t*c5))));

   /*----------------------------------------------------*/
   /* for a single call, better to factor the polynomial */
   /*----------------------------------------------------*/
   f0 = 1.0 + t;
   f2 = t*t;
   f4 = f2*f2;
   f1 = c2 + t*c3;
   f3 = c4 + t*c5;

   poly = (f0 + f1*f2) + f3*f4;

   /*---------------------------------*/
   /* construct the result and return */
   /*---------------------------------*/
   result = hfactor* poly;

   /*-----------------------------------*/
   /* check input value for valid range */
   /*-----------------------------------*/
   if      (x < -709.0) return 0.0;
   else if (x >  709.0) return HUGE_VAL;
   else return result;
}

// TRACING =================================================

#include <time.h>

double timediff(
    struct timespec* t1,
    struct timespec* t0)
{
  struct timespec dt;

  if (t1->tv_nsec >= t0->tv_nsec) {
    dt.tv_sec  = t1->tv_sec  - t0->tv_sec;
    dt.tv_nsec = t1->tv_nsec - t0->tv_nsec;
  }
  else {
    dt.tv_sec  = t1->tv_sec  - t0->tv_sec  - 1;
    dt.tv_nsec = t1->tv_nsec - t0->tv_nsec + 1E+9;
  }

  return (double) dt.tv_sec + (double) dt.tv_nsec / 1.0E+9;
}

// =========================================================

int main(int argc, char * argv[])
{
  int i, k, exchbytes, nranks, ch, help_flag = 0;
  int iter, maxiter, calib_iters, bin, numbins, method;
  long npts;
  double  ssum, data_volume, bw;
  double  t1, t2, /*t3,*/ tmin, tmax;
  double * xrand, * table, * block_data;
  double * dev_xrand, * dev_table, * dev_block_data;
  double mean, * alldev, * allavg;
  double minavg, maxavg;
  int minrank, maxrank;
  unsigned long seed;
  int nrand = 1000, ntable = 256;
  int local_rank, ranks_per_node;
  MPI_Comm local_comm;

  // TRACING
  int tracing = 0;
  double threshold = INFINITY;

  double * tcomm, * tcomp, * tstep;
  double compute_interval_msec;
  double elapsed1, elapsed2, target_measurement_time;

  double histo_bin_width, xhisto, hmin, prob;
  double tavg, ssq, samples, relative_variation;
  double dev, sigma, scb, sp4, skewness, kurtosis;
  int * histo;

  int jobid, rank, green_light, dump_data = 0, tag = 99;
  float * compbuf, * stepbuf, * floatcomp, * floatstep;
  char outfile[80];
  MPI_Status status;

  char host[80], * ptr, * snames, * rnames;
  int * sort_key;
  int num_nodes, color, key;
  double * compmax, compmin, compavg, compssq;
  double sigma_comp, samples_total;
  double * aggregate_sigma_comp, * aggregate_compavg;
  time_t current_time;
  char * time_string;
  FILE * ofp;
  MPI_Comm node_comm;

  int adjust_freq = 0;
  nvmlDevice_t nvmldevice, * device;
  unsigned int device_count, graphicsClockMHz;
  int numDevices, myDevice, gpu;

  hipEvent_t cudaBeg, cudaEnd;
  //float kernel_msec;
  int kernel_timing;

  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
  MPI_Comm_size(MPI_COMM_WORLD, &nranks);
  get_local_communicator(&ranks_per_node, &local_rank, &local_comm);

  MPI_Barrier(MPI_COMM_WORLD);
  current_time = time(NULL);
  time_string = ctime(&current_time);
  if (myrank == 0) printf("starting time : %s\n", time_string);

  // each MPI rank selects a device
  CUDA_RC(hipGetDeviceCount(&numDevices));
  myDevice = myrank % numDevices;
  CUDA_RC(hipSetDevice(myDevice));

  if (myrank == 0) printf("checking program args ...\n");

  // set sensible default values then check program args
  numbins = 50;
  compute_interval_msec = 50.0;
  target_measurement_time = 50.0;  // units of seconds
  exchbytes = 100;
  method = EXCHANGE;
  kernel_timing = 0;

  while (1) {
     ch = getopt(argc, argv, "hdkf:m:c:t:n:x:j:");
     if (ch == -1) break;
     switch (ch) {
        case 'h':
           help_flag = 1;
           break;
        case 'c':  // set the compute interval in msec
           compute_interval_msec = (double) atoi(optarg);
           break;
        case 'm':  // choose the comunication method : barrier or exchange
           if (0 == strncasecmp(optarg, "barrier", 7)) method = BARRIER;
           else                                        method = EXCHANGE;
           break;
        case 't':  // set the target measurement time in sec
           target_measurement_time = atoi(optarg);
           break;
        case 'f':  // set the GPU frequency
           adjust_freq = 1;
           graphicsClockMHz = atoi(optarg);
           break;
        case 'n':  // set the number of histogram bins
           numbins = atoi(optarg);
           break;
        case 'x':  // set the number of bytes for neighbor exchange
           exchbytes = atoi(optarg);
           break;
        /*
        case 'k':  // optionally use kernel time for tcomp
           kernel_timing = 1;
           break;
        */
        case 'j':  // set jitter tracing threshold value in msec
           tracing = 1;
           threshold = atof(optarg) / 1000.0;
           break;
        case 'd':  // optionally dump all timing data
           dump_data = 1;
           break;
        default:
           break;
     }
  }

  if (help_flag) {
     if (myrank == 0) print_help();
     MPI_Finalize();
     return 0;
  }

  maxiter = (int) (1.0e3 * target_measurement_time / compute_interval_msec );

  if (kernel_timing) {
     CUDA_RC(hipEventCreate(&cudaBeg));
     CUDA_RC(hipEventCreate(&cudaEnd));
     if (myrank == 0) printf("compute times are cuda kernel times, not counting memcpy\n");
  }
  else {
     if (myrank == 0) printf("compute times include cuda kernel plus memcpy times\n");
  }

  if (NVML_SUCCESS != nvmlInit()) {
     fprintf(stderr, "failed to initialize NVML ... exiting\n");
     MPI_Abort(MPI_COMM_WORLD, 1);
  }

  if (NVML_SUCCESS != nvmlDeviceGetCount(&device_count)) {
     fprintf(stderr, "nvmlDeviceGetCount failed ... exiting\n");
     MPI_Abort(MPI_COMM_WORLD, 1);
  }

  device = (nvmlDevice_t *) malloc(device_count*sizeof(nvmlDevice_t));

  for (i = 0; i < device_count; i++) {
     if (NVML_SUCCESS != nvmlDeviceGetHandleByIndex(i, &device[i])) {
        fprintf(stderr, "nvmlDeviceGetHandleByIndex failed ... exiting\n");
        MPI_Abort(MPI_COMM_WORLD, 1);
     }
  }

  // for use in nvml queries
  nvmldevice = device[myDevice];

  if (adjust_freq) {
     if (NVML_SUCCESS != nvmlDeviceSetApplicationsClocks ( nvmldevice, 877, graphicsClockMHz )) {
        fprintf(stderr, "rank %d failed to set the GPU freq to %d for device %d ... continuing\n", myrank, graphicsClockMHz, myDevice);
     }
  }

  if (myrank == 0) {
     if (method == EXCHANGE) {
        printf("using neighbor exchange : compute_interval_msec = %.2lf, target measurement time = %.1lf sec, numbins = %d, exchbytes = %d\n",
                compute_interval_msec, target_measurement_time, numbins, exchbytes);
     }
     else {
        printf("using global barrier : compute_interval_msec = %.2lf, target measurement time = %.1lf sec, numbins = %d\n",
                compute_interval_msec, target_measurement_time, numbins);
     }
  }

  npex = (int) (0.1 + sqrt((double) nranks));
  while (nranks % npex != 0) npex++;
  npey = nranks / npex;

  exchcount = exchbytes / sizeof(double);

  mycol = myrank % npex;
  myrow = myrank / npex;

  if (myrank == 0) printf("starting with nranks = %d\n", nranks);

  if (myrank == 0 && method == EXCHANGE) printf("using npex = %d, npey = %d\n", npex, npey);

  histo = (int *) malloc(numbins*sizeof(int));
  tcomm = (double *) malloc(maxiter*sizeof(double));
  tcomp = (double *) malloc(maxiter*sizeof(double));
  tstep = (double *) malloc(maxiter*sizeof(double));

  sendn = (double *) malloc(exchbytes);
  sends = (double *) malloc(exchbytes);
  sende = (double *) malloc(exchbytes);
  sendw = (double *) malloc(exchbytes);
  recvn = (double *) malloc(exchbytes);
  recvs = (double *) malloc(exchbytes);
  recve = (double *) malloc(exchbytes);
  recvw = (double *) malloc(exchbytes);

  for (i = 0; i< exchcount; i++) sendn[i] = (double) myrank;
  for (i = 0; i< exchcount; i++) sends[i] = (double) myrank;
  for (i = 0; i< exchcount; i++) sende[i] = (double) myrank;
  for (i = 0; i< exchcount; i++) sendw[i] = (double) myrank;

  for (i = 0; i< exchcount; i++) recvn[i] = 0.0;
  for (i = 0; i< exchcount; i++) recvs[i] = 0.0;
  for (i = 0; i< exchcount; i++) recve[i] = 0.0;
  for (i = 0; i< exchcount; i++) recvw[i] = 0.0;

  xrand = (double *) malloc(nrand*sizeof(double));

  seed = 13579UL;

  TMrand(seed, nrand, xrand);

  table = (double *) malloc(ntable*sizeof(double));

  // make a copy of the exp look-up table
  for (i = 0; i < ntable; i++) table[i] = exp_table[i];

  // allocate GPU data
  CUDA_RC(hipMalloc((void **)&dev_table, ntable*sizeof(double)));
  CUDA_RC(hipMalloc((void **)&dev_xrand, nrand*sizeof(double)));
  CUDA_RC(hipMalloc((void **)&dev_block_data, MAX_BLOCKS*sizeof(double)));

  block_data = (double *) malloc(MAX_BLOCKS*sizeof(double));

  // copy data to the GPU one time
  CUDA_RC(hipMemcpy(dev_table, table, ntable*sizeof(double), hipMemcpyHostToDevice));
  CUDA_RC(hipMemcpy(dev_xrand, xrand, nrand*sizeof(double),  hipMemcpyHostToDevice));

  // this initial guess is pretty good for Volta ... adjust if necessary
  npts = (long) ( 1.6e10 * compute_interval_msec / 125.0 );
  ssum = 0.0;

  if (myrank == 0) printf("initial npts = %ld\n", npts);

  calib_iters = (int) ( 200.0 * 50.0 / compute_interval_msec );

  MPI_Barrier(MPI_COMM_WORLD);

  // make one call that is not timed
  int threadsPerBlock = THREADS_PER_BLOCK;
  int numBlocks = (npts + threadsPerBlock - 1) / threadsPerBlock;
  if (numBlocks > MAX_BLOCKS) numBlocks = MAX_BLOCKS;
  compute<<<numBlocks, threadsPerBlock>>>(npts, nrand, dev_xrand, dev_table, dev_block_data);
  CUDA_RC(hipMemcpy(block_data, dev_block_data, numBlocks*sizeof(double), hipMemcpyDeviceToHost));
  for (int j = 0; j < numBlocks; j++)  ssum += block_data[j];

  ssum += 1.0;

  if (method == EXCHANGE) exch(0);
  else                    MPI_Barrier(MPI_COMM_WORLD);

  if (myrank == 0) printf("calibrating compute time ..\n");

  MPI_Barrier(MPI_COMM_WORLD);

  // time calls to the compute routine and adjust npts
  // compute for long enough to ramp up power
  t1 = MPI_Wtime();
  for (i = 0; i < calib_iters ; i++) {
    if (kernel_timing) CUDA_RC(hipEventRecord(cudaBeg));
    compute<<<numBlocks, threadsPerBlock>>>(npts, nrand, dev_xrand, dev_table, dev_block_data);
    if (kernel_timing) CUDA_RC(hipEventRecord(cudaEnd));
    CUDA_RC(hipMemcpy(block_data, dev_block_data, numBlocks*sizeof(double), hipMemcpyDeviceToHost));
    for (int j = 0; j < numBlocks; j++)  ssum += block_data[j];
    ssum += 1.0;
  }
  t2 = MPI_Wtime();
  tmin = (t2 - t1)/((double) calib_iters);

  if (myrank == 0) printf("initial tmin = %.2lf msec\n", 1.0e3*tmin);

  // find the global minimum time and use that as the reference
  MPI_Allreduce(MPI_IN_PLACE, &tmin, 1, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);

  // reset npts using measured compute time
  npts = (long) (((double) npts)*compute_interval_msec/(1.0e3*tmin));

  if (myrank == 0) printf("first iteration : npts = %ld\n", npts);

  numBlocks = (npts + threadsPerBlock - 1) / threadsPerBlock;
  if (numBlocks > MAX_BLOCKS) numBlocks = MAX_BLOCKS;

  // repeat this process one more time
  MPI_Barrier(MPI_COMM_WORLD);
  t1 = MPI_Wtime();
  for (i = 0; i < calib_iters ; i++) {
    if (kernel_timing) CUDA_RC(hipEventRecord(cudaBeg));
    compute<<<numBlocks, threadsPerBlock>>>(npts, nrand, dev_xrand, dev_table, dev_block_data);
    if (kernel_timing) CUDA_RC(hipEventRecord(cudaEnd));
    CUDA_RC(hipMemcpy(block_data, dev_block_data, numBlocks*sizeof(double), hipMemcpyDeviceToHost));
    for (int j = 0; j < numBlocks; j++)  ssum += block_data[j];
    ssum += 1.0;
  }
  t2 = MPI_Wtime();
  tmin = (t2 - t1)/((double) calib_iters);

  // find the global minimum time and use that as the reference
  MPI_Allreduce(MPI_IN_PLACE, &tmin, 1, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);

  // reset npts using measured compute time
  npts = (long) (((double) npts)*compute_interval_msec/(1.0e3*tmin));

  if (myrank == 0) printf("using npts = %ld, tmin = %.2lf msec\n", npts, 1.0e3*tmin);

  numBlocks = (npts + threadsPerBlock - 1) / threadsPerBlock;
  if (numBlocks > MAX_BLOCKS) numBlocks = MAX_BLOCKS;

  compute_interval_msec = 1.0e3*tmin;

  // TRACING
  struct timespec* T1 = (struct timespec*) malloc(maxiter * sizeof(struct timespec));
  struct timespec* T2 = (struct timespec*) malloc(maxiter * sizeof(struct timespec));
  struct timespec* T3 = (struct timespec*) malloc(maxiter * sizeof(struct timespec));

  MPI_Barrier(MPI_COMM_WORLD);

  MPI_Pcontrol(1);

  elapsed1 = MPI_Wtime();

  //==============================================
  // time a number of {compute, communicate} steps
  //==============================================

  for (iter = 0; iter < maxiter; iter++) {
    if ( (iter + 1) % 100 == 0) MPI_Barrier(MPI_COMM_WORLD);

clock_gettime(CLOCK_MONOTONIC, &T1[iter]);
    compute<<<numBlocks, threadsPerBlock>>>(npts, nrand, dev_xrand, dev_table, dev_block_data);
    CUDA_RC(hipMemcpy(block_data, dev_block_data, numBlocks*sizeof(double), hipMemcpyDeviceToHost));
clock_gettime(CLOCK_MONOTONIC, &T2[iter]);
    if (method == EXCHANGE)     exch(iter);
    else                        MPI_Barrier(MPI_COMM_WORLD);
clock_gettime(CLOCK_MONOTONIC, &T3[iter]);
  }

  /*
  for (iter = 0; iter < maxiter; iter++) {
    if ( (iter + 1) % 100 == 0) MPI_Barrier(MPI_COMM_WORLD);
    t1 = MPI_Wtime();
    if (kernel_timing) CUDA_RC(hipEventRecord(cudaBeg));
    compute<<<numBlocks, threadsPerBlock>>>(npts, nrand, dev_xrand, dev_table, dev_block_data);
    if (kernel_timing) CUDA_RC(hipEventRecord(cudaEnd));
    CUDA_RC(hipMemcpy(block_data, dev_block_data, numBlocks*sizeof(double), hipMemcpyDeviceToHost));
    if (kernel_timing) CUDA_RC(hipEventSynchronize(cudaEnd));
    for (int j = 0; j < numBlocks; j++)  ssum += block_data[j];
    ssum += 1.0;
    t2 = MPI_Wtime();
    if (method == EXCHANGE) exch(iter);
    else                    MPI_Barrier(MPI_COMM_WORLD);
    t3 = MPI_Wtime();
    if (kernel_timing) {
       CUDA_RC(hipEventElapsedTime(&kernel_msec, cudaBeg, cudaEnd));
       tcomp[iter] = 1.0e-3 * ((double) kernel_msec);
    }
    else tcomp[iter] = t2 - t1;
    tcomm[iter] = t3 - t2;
    tstep[iter] = t3 - t1;
  }
  */

  MPI_Allreduce(MPI_IN_PLACE, tcomm, maxiter, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

  elapsed2 = MPI_Wtime();

  MPI_Pcontrol(0);

  for (iter = 0; iter < maxiter; iter++) {
    tcomp[iter] = timediff(&T2[iter], &T1[iter]);
    tcomm[iter] = timediff(&T3[iter], &T2[iter]);
    tstep[iter] = timediff(&T3[iter], &T1[iter]);
  }

  // compute mean and relative variation rank by rank
  mean = 0.0;
  for (iter = 0; iter < maxiter; iter++) mean += tcomp[iter];
  mean = mean / ((double) maxiter);

  ssq = 0.0;
  for (iter = 0; iter < maxiter; iter++) ssq += (tcomp[iter] - mean) * (tcomp[iter] - mean);

  sigma = sqrt(ssq/((double) maxiter));

  relative_variation = 100.0*sigma/mean;

  alldev = (double *) malloc(nranks*sizeof(double));
  allavg = (double *) malloc(nranks*sizeof(double));

  MPI_Gather(&relative_variation,     1, MPI_DOUBLE, alldev,  1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Gather(&mean,                   1, MPI_DOUBLE, allavg,  1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

  // max data volume per rank per step ... n,e,s,w both send and recv = 8x
  data_volume = 8.0*((double) exchbytes);

  tmax = 0.0; tmin = 1.0e30;
  for (iter = 0; iter < maxiter; iter++) {
    if (tcomm[iter] > tmax) tmax = tcomm[iter];
    if (tcomm[iter] < tmin) tmin = tcomm[iter];
  }

  if (myrank == 0) printf("max communication time in msec = %.3lf\n",  1.0e3*tmax);

  // histogram the data for max communication time per iteration
  histo_bin_width = (tmax - tmin)/((double) (numbins - 1));

  hmin = tmin - 0.5*histo_bin_width;

  for (bin = 0; bin < numbins; bin++) histo[bin] = 0;

  for (iter = 0; iter < maxiter; iter++) {
    if (histo_bin_width > 0.0)  bin = (int) ((tcomm[iter] - hmin)/histo_bin_width);
    else                        bin = 0;
    if ((bin >= 0) && (bin < numbins)) histo[bin]++;
  }

  if (myrank == 0) {
    printf("\n");
    printf("histogram of max communication times per step\n");
    printf("      msec       count\n");
    for (bin = 0; bin < numbins; bin++) {
      xhisto = tmin + histo_bin_width*((double) bin);
      printf("%10.3lf  %10d\n", xhisto*1.0e3, histo[bin]);
    }
  }

  // for per-node analysis, focus on the compute times
  compmax = (double *) malloc(maxiter*sizeof(double));

  // find the global minimum computation time over all iterations
  compmin = 1.0e30;
  for (iter = 0; iter < maxiter; iter++) {
    if (tcomp[iter] < compmin) compmin = tcomp[iter];
  }

  MPI_Allreduce(MPI_IN_PLACE, &compmin, 1, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);

  // find the max compute time for each iteration on a per-node basis
  MPI_Allreduce(tcomp, compmax, maxiter, MPI_DOUBLE, MPI_MAX, local_comm);

  // find the global avg time for computation
  compavg = 0.0;
  for (iter = 0; iter < maxiter; iter++) compavg += tcomp[iter];

  MPI_Allreduce(MPI_IN_PLACE, &compavg, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);

  samples_total = ((double) maxiter) * ((double) nranks);

  compavg = compavg / samples_total;

  // look at the max computation times per iteration for analysis by node
  compssq = 0.0;
  for (iter = 0; iter < maxiter; iter++) {
    dev = compmax[iter] - compavg;
    compssq += dev*dev;
  }

  sigma_comp = sqrt(compssq/((double) maxiter));

  num_nodes = nranks / ranks_per_node;

  color = local_rank;
  key = myrank;
  MPI_Comm_split(MPI_COMM_WORLD, color, key, &node_comm);

  if (local_rank == 0) {

     aggregate_sigma_comp = (double *) malloc(num_nodes*sizeof(double));
     aggregate_compavg    = (double *) malloc(num_nodes*sizeof(double));

     MPI_Allgather(&sigma_comp, 1, MPI_DOUBLE, aggregate_sigma_comp, 1, MPI_DOUBLE, node_comm);
     MPI_Allgather(&compavg,    1, MPI_DOUBLE, aggregate_compavg,    1, MPI_DOUBLE, node_comm);

     snames = (char *) malloc(num_nodes*sizeof(host));
     rnames = (char *) malloc(num_nodes*sizeof(host));
     sort_key = (int *) malloc(num_nodes*sizeof(int));

     gethostname(host, sizeof(host));

     for (i=0; i<sizeof(host); i++) {
        if (host[i] == '.') {
           host[i] = '\0';
           break;
        }
     }

     for (i=0; i<num_nodes; i++)  {
       ptr = snames + i*sizeof(host);
       strncpy(ptr, host, sizeof(host));
     }

     MPI_Alltoall(snames, sizeof(host), MPI_BYTE, rnames, sizeof(host), MPI_BYTE, node_comm);

     // sort by node-local sigma for computation times
     // aggregate_sigma_comp is returned in sorted order, along with the sort key
     sortx(aggregate_sigma_comp, num_nodes, sort_key, SORT_DESCENDING_ORDER);

     if (myrank == 0) {
        printf("\n");
        printf(" percent variation = 100*sigma/mean for computation times by node:\n");
        printf("              host    percent variation\n");
        for (i=0; i< num_nodes; i++) {
           k = sort_key[i];
           ptr = rnames + k*sizeof(host);
           relative_variation = 100.0*aggregate_sigma_comp[i] / aggregate_compavg[k];
           printf("%18s    %10.3lf\n", ptr, relative_variation);
        }
        printf("\n");
     }
  }


  // analyze all step time data for all ranks
  tmin = 1.0e30; tmax = 0.0;
  for (iter = 0; iter < maxiter; iter++) {
     if (tstep[iter] > tmax) tmax = tstep[iter];
     if (tstep[iter] < tmin) tmin = tstep[iter];
  }

  if (method == BARRIER) {
     // in this case there is just one step time per iteration, the same for all ranks
     tavg = (elapsed2 - elapsed1) / ((double) maxiter);

     // compute moments for the time-step samples
     ssq = 0.0; scb = 0.0; sp4 = 0.0;
     for (iter = 0; iter < maxiter; iter++) {
       dev = tstep[iter] - tavg;
       ssq += dev*dev;
       scb += dev*dev*dev;
       sp4 += dev*dev*dev*dev;
     }

     samples = ((double) maxiter);
  }
  else {
     // analyze all step time data for every rank
     tavg = 0.0;
     for (iter = 0; iter < maxiter; iter++) tavg += tstep[iter];

     // global avg for step times
     MPI_Allreduce(MPI_IN_PLACE, &tavg, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);

     samples = ((double) maxiter) * ((double) nranks);

     tavg = tavg / samples;

     // compute moments for the time-step samples ... per rank
     ssq = 0.0; scb = 0.0; sp4 = 0.0;
     for (iter = 0; iter < maxiter; iter++) {
       dev = tstep[iter] - tavg;
       ssq += dev*dev;
       scb += dev*dev*dev;
       sp4 += dev*dev*dev*dev;
     }

     // compute global statistics on the step times
     MPI_Allreduce(MPI_IN_PLACE, &ssq, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
     MPI_Allreduce(MPI_IN_PLACE, &scb, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
     MPI_Allreduce(MPI_IN_PLACE, &sp4, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  }

  sigma = sqrt(ssq/samples);

  relative_variation = sigma / tavg;

  skewness = scb / (samples * sigma * sigma * sigma);

  kurtosis = sp4 / (samples * sigma * sigma * sigma * sigma);

  // find the global min and max step times
  MPI_Allreduce(MPI_IN_PLACE, &tmin, 1, MPI_DOUBLE, MPI_MIN, MPI_COMM_WORLD);
  MPI_Allreduce(MPI_IN_PLACE, &tmax, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

  histo_bin_width = (tmax - tmin)/((double) (numbins - 1));
  hmin = tmin - 0.5*histo_bin_width;

  for (bin = 0; bin < numbins; bin++) histo[bin] = 0;

  // each rank histograms its own data
  for (iter = 0; iter < maxiter; iter++) {
    if (histo_bin_width > 0.0)  bin = (int) ((tstep[iter] - hmin)/histo_bin_width);
    else                        bin = 0;
    if ((bin >= 0) && (bin < numbins)) histo[bin]++;
  }

  // for the exchange method, we should sum over all MPI ranks
  if (method == EXCHANGE) {
     MPI_Allreduce(MPI_IN_PLACE, histo, numbins, MPI_INT, MPI_SUM, MPI_COMM_WORLD);
  }

  if (myrank == 0) {
    printf("histogram of step times for all ranks\n");
    printf("      msec       count               density\n");
    for (bin = 0; bin < numbins; bin++) {
      xhisto = tmin + histo_bin_width*((double) bin);
      prob = 1.0e-3*((double) histo[bin]) / histo_bin_width;
      printf("%10.3lf  %10d  %20.4lf\n", xhisto*1.0e3, histo[bin], prob);
    }
    printf("\n");
  }

  if (myrank == 0) {
    bw = 1.0e-9*0.5*((double) ranks_per_node)*data_volume/(tavg - compmin);
    printf("min computational interval in msec = %.4lf\n", 1.0e3*compmin);
    printf("the average time per step in msec  = %.4lf\n", 1.0e3*tavg);
    printf("percent variation in step times    = %.3lf\n", 100.0*relative_variation);
    printf("skewness of step time distribution = %.3le\n", skewness);
    printf("kurtosis of step time distribution = %.3le\n", kurtosis);
    printf("elapsed time (sec)                 = %.3lf\n", elapsed2 - elapsed1);
    printf("measured time / target time        = %.3lf\n", (elapsed2 - elapsed1)/target_measurement_time);
    printf("effective parallel efficiency      = %.3lf\n", target_measurement_time/(elapsed2 - elapsed1));
    if (method == EXCHANGE) printf("effective exch bw per node         = %.3le GB/sec\n", bw);
    printf("ssum =  %.6le\n\n", ssum);
  }

  if (myrank == 0) {
    maxavg = 0.0;
    minavg = 1.0e30;
    maxrank = 0;
    minrank = 0;
    for (i=0; i<nranks; i++) {
      if (allavg[i] > maxavg)  {
        maxavg = allavg[i];
        maxrank = i;
      }
      if (allavg[i] < minavg)  {
        minavg = allavg[i];
        minrank = i;
      }
    }
    printf("\n");
    printf("max avg compute interval = %.3lf msec for rank %d\n", 1.0e3*maxavg, maxrank);
    printf("min avg compute interval = %.3lf msec for rank %d\n", 1.0e3*minavg, minrank);
    printf("\n");
    printf("mean compute interval (msec) and percent relative variation by rank:\n");
    printf("  rank      host   gpu  <compute(msec)> %%variation\n");
    for (i=0; i<nranks; i++) {
       k = i / numDevices;
       ptr = rnames + k*sizeof(host);
       gpu = i % numDevices;
       printf("%6d %10s %4d %12.3lf  %10.2lf\n", i, ptr, gpu, 1.0e3*allavg[i], alldev[i]);
    }
    printf("\n");
  }

  if (dump_data) {
    floatcomp = (float *) malloc(maxiter*sizeof(float));
    floatstep = (float *) malloc(maxiter*sizeof(float));
    for (iter=0; iter<maxiter; iter++) {
      floatcomp[iter] = (float) tcomp[iter];
      floatstep[iter] = (float) tstep[iter];
    }
    if (myrank == 0) {
      compbuf = (float *) malloc(maxiter*sizeof(float));
      stepbuf = (float *) malloc(maxiter*sizeof(float));
      ptr = getenv("LSB_JOBID");
      if (ptr == NULL) jobid = getpid();
      else             jobid = atoi(ptr);
      sprintf(outfile, "%d.timing_data", jobid);
      ofp = fopen(outfile, "w");
      if (ofp != NULL) {
        fwrite(floatcomp, sizeof(float), maxiter, ofp);
        fwrite(floatstep, sizeof(float), maxiter, ofp);
      }
      for (rank=1; rank<nranks; rank++) {
         MPI_Send(&green_light, 1, MPI_INT, rank, tag, MPI_COMM_WORLD);
         MPI_Recv(compbuf, maxiter, MPI_FLOAT, rank, tag, MPI_COMM_WORLD, &status);
         MPI_Recv(stepbuf, maxiter, MPI_FLOAT, rank, tag, MPI_COMM_WORLD, &status);
         if (ofp != NULL) {
           fwrite(compbuf, sizeof(float), maxiter, ofp);
           fwrite(stepbuf, sizeof(float), maxiter, ofp);
         }
      }
    }
    else {
      MPI_Recv(&green_light, 1, MPI_INT, 0, tag, MPI_COMM_WORLD, &status);
      MPI_Send(floatcomp, maxiter, MPI_FLOAT, 0, tag, MPI_COMM_WORLD);
      MPI_Send(floatstep, maxiter, MPI_FLOAT, 0, tag, MPI_COMM_WORLD);
    }
  }

  //
  // TRACING
  // 

  if (tracing) {
    char hostname[64] = {0};
    gethostname(hostname, 64);
    int cpu;
    cpu_set_t set;
    sched_getaffinity(0, sizeof(set), &set);
    int j;
    for (j = 0; j < CPU_SETSIZE; ++j) {
      if (CPU_ISSET(j, &set)) {
        cpu = j;    
        break;
      }
    }

    if (myrank == 0) {
      struct timespec* T1_buf = (struct timespec*) malloc(maxiter * sizeof(struct timespec));
      struct timespec* T2_buf = (struct timespec*) malloc(maxiter * sizeof(struct timespec));
      struct timespec* T3_buf = (struct timespec*) malloc(maxiter * sizeof(struct timespec));
      memcpy(T1_buf, T1, maxiter * sizeof(struct timespec));
      memcpy(T2_buf, T2, maxiter * sizeof(struct timespec));
      memcpy(T3_buf, T3, maxiter * sizeof(struct timespec));

      char rank_host[64] = {0};
      memcpy(rank_host, hostname, 64);
      int rank_cpu;
      rank_cpu = cpu;

      char tracing_fn[256] = {0};
      sprintf(tracing_fn, "job%s.jtf", getenv("LSB_JOBID"));
      FILE* tracing_fp = fopen(tracing_fn, "w");

      for (rank = 0; rank < nranks; rank++) {
        if (rank) {
          MPI_Recv(rank_host, 64, MPI_BYTE, rank, tag, MPI_COMM_WORLD, &status);
          MPI_Recv(&rank_cpu, 1,  MPI_INT,  rank, tag, MPI_COMM_WORLD, &status);
          MPI_Recv(T1_buf, maxiter * sizeof(struct timespec), MPI_BYTE, rank, tag, MPI_COMM_WORLD, &status);
          MPI_Recv(T2_buf, maxiter * sizeof(struct timespec), MPI_BYTE, rank, tag, MPI_COMM_WORLD, &status);
          MPI_Recv(T3_buf, maxiter * sizeof(struct timespec), MPI_BYTE, rank, tag, MPI_COMM_WORLD, &status);
        }

        printf("[jtf] rank=%d host=%s cpu=%d maxiter=%d\n", rank, rank_host, rank_cpu, maxiter);

        for (iter = 0; iter < maxiter; iter++) {
          double dt = timediff(&T2_buf[iter], &T1_buf[iter]);
          if (dt >= threshold) {
            fprintf(tracing_fp, 
                "%-10d %s %-3d %10ld.%06ld %10ld.%06ld %16.6lf\n",
                rank,
                rank_host,
                rank_cpu,
                T1_buf[iter].tv_sec, T1_buf[iter].tv_nsec / 1000,
                T2_buf[iter].tv_sec, T2_buf[iter].tv_nsec / 1000,
                dt);
          } // threshold
        } // iter loop
      } // rank loop
    } // rank 0
    else {
      MPI_Send(hostname, 64, MPI_BYTE, 0, tag, MPI_COMM_WORLD);
      MPI_Send(&cpu,     1,  MPI_INT,  0, tag, MPI_COMM_WORLD);
      MPI_Send(T1, maxiter * sizeof(struct timespec), MPI_BYTE, 0, tag, MPI_COMM_WORLD);
      MPI_Send(T2, maxiter * sizeof(struct timespec), MPI_BYTE, 0, tag, MPI_COMM_WORLD);
      MPI_Send(T3, maxiter * sizeof(struct timespec), MPI_BYTE, 0, tag, MPI_COMM_WORLD);
    } // rank x
  } // tracing

  MPI_Finalize();

  return 0;
}


// -----------------------------------
// routine for boundary exchange
// -----------------------------------
void exch(int iter)
{
  int ireq, tag, north, south, east, west;
  MPI_Request req[8];
  MPI_Status status[8];

  ireq = 0;

  tag = 99 + 20*iter;

  // exchange data with partner to the north
  if (myrow != (npey - 1)) {
    north = myrank + npex;
    MPI_Irecv(recvn, exchcount, MPI_DOUBLE, north, tag, MPI_COMM_WORLD, &req[ireq]);
    ireq = ireq + 1;
    MPI_Isend(sendn, exchcount, MPI_DOUBLE, north, tag, MPI_COMM_WORLD, &req[ireq]);
    ireq = ireq + 1;
  }

  // exchange data with partner to the east
  if (mycol != (npex - 1)) {
    east = myrank + 1;
    MPI_Irecv(recve, exchcount, MPI_DOUBLE, east, tag, MPI_COMM_WORLD, &req[ireq]);
    ireq = ireq + 1;
    MPI_Isend(sende, exchcount, MPI_DOUBLE, east, tag, MPI_COMM_WORLD, &req[ireq]);
    ireq = ireq + 1;
  }

  // exchange data with partner to the south
  if (myrow != 0) {
    south = myrank - npex;
    MPI_Irecv(recvs, exchcount, MPI_DOUBLE, south, tag, MPI_COMM_WORLD, &req[ireq]);
    ireq = ireq + 1;
    MPI_Isend(sends, exchcount, MPI_DOUBLE, south, tag, MPI_COMM_WORLD, &req[ireq]);
    ireq = ireq + 1;
  }

  // exchange data with partner to the west
  if (mycol != 0) {
    west = myrank - 1;
    MPI_Irecv(recvw, exchcount, MPI_DOUBLE, west, tag, MPI_COMM_WORLD, &req[ireq]);
    ireq = ireq + 1;
    MPI_Isend(sendw, exchcount, MPI_DOUBLE, west, tag, MPI_COMM_WORLD, &req[ireq]);
    ireq = ireq + 1;
  }

  MPI_Waitall(ireq, req, status);

  return;

}

// ----------------------------------------------------------
// routine for computation; data is returned in the out array
// which must be copied back to the host for final reduction
// ----------------------------------------------------------
__global__ void compute(long npts, int nrand, double * xrand, double * table, double * out)
{
  double tsum, y;
  long lrand, ndx;
  __shared__ double acc[NUM_WARPS];
  int lane   = threadIdx.x % WARP_SIZE;
  int warp   = threadIdx.x / WARP_SIZE;

  lrand = (long) nrand;

  tsum = 0.0;
  for (long i = blockDim.x * blockIdx.x + threadIdx.x;  i < npts; i += blockDim.x * gridDim.x) {
    ndx = i % lrand;
    y = lutexp(xrand[ndx], table);
    tsum += y;
  }

  // reduce over threads in a warp
  for (int shift = WARP_SIZE/2; shift > 0; shift /= 2) tsum += __shfl_down_sync(0xffffffff, tsum, shift);

  // save values for this thread block in shared memory
  if (lane == 0) acc[warp] = tsum;
  __syncthreads();

  // reduce once more to get a value per thread block
  if (warp == 0) {
     tsum = (lane < NUM_WARPS) ? acc[lane] : 0.0;
     for (int shift = NUM_WARPS/2; shift > 0; shift /= 2) tsum += __shfl_down_sync(0xffffffff, tsum, shift);
  }

  // save the per-block values for final reduction on the host
  if (threadIdx.x == 0) out[blockIdx.x] = tsum;

}

// -----------------------------------
// help message
// -----------------------------------
void print_help(void)
{
   printf("Syntax: mpirun -np #ranks osnoise [-d] [-c compute_interval_msec] [-t target_measurement_time] [-n numbins] [-x exchbytes] [-m method]\n");
   printf("  -d to dump data\n"
          "  -t in units of sec\n"
          "  -x in units of bytes\n");
}


void get_local_communicator(int * ppn, int * prank, MPI_Comm * pcomm)
{
  int i, myrank, nranks;
  char * ptr, * snames, * rnames, host[80];
  int match, color;
  int local_rank, ranks_per_node;
  MPI_Comm local_comm;

  MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
  MPI_Comm_size(MPI_COMM_WORLD, &nranks);

  // make a communicator of all ranks on this node
  snames = (char *) malloc(nranks*sizeof(host));
  rnames = (char *) malloc(nranks*sizeof(host));
  gethostname(host, sizeof(host));

  for (i=0; i<sizeof(host); i++) {
     if (host[i] == '.') {
         host[i] = '\0';
         break;
     }
  }

  for (i=0; i<nranks; i++)  {
    ptr = snames + i*sizeof(host);
    strncpy(ptr, host, sizeof(host));
  }

  MPI_Alltoall(snames, sizeof(host), MPI_BYTE,
               rnames, sizeof(host), MPI_BYTE, MPI_COMM_WORLD);
  color = 0;
  match = 0;
  for (i=0; i<nranks; i++) {
    ptr = rnames + i*sizeof(host);
    if (strcmp(host, ptr) == 0) {
      match++;
      if (match == 1) color = i;
    }
  }

  MPI_Comm_split(MPI_COMM_WORLD, color, myrank, &local_comm);
  MPI_Comm_rank(local_comm, &local_rank);
  MPI_Comm_size(local_comm, &ranks_per_node);

  *ppn = ranks_per_node;
  *prank = local_rank;
  *pcomm = local_comm;

  return;
}

//===========================================================================
// incremental Shell sort with increment array: inc[k] = 1 + 3*2^k + 4^(k+1)
//===========================================================================
void sortx(double * arr , int n, int * ind, int flag)
{
   int h, i, j, k, inc[20];
   int numinc, pwr2, pwr4;
   double val;

   if (n <= 1) {
      ind[0] = 0;
      return;
   }

   pwr2 = 1;
   pwr4 = 4;

   numinc = 0;
   h = 1;
   inc[numinc] = h;
   while (numinc < 20) {
      h = 1 + 3*pwr2 + pwr4;
      if (h > n) break;
      numinc++;
      inc[numinc] = h;
      pwr2 *= 2;
      pwr4 *= 4;
   }

   for (i=0; i<n; i++) ind[i] = i;

   if (flag > 0) { // sort in increasing order
      for (; numinc >= 0; numinc--) {
         h = inc[numinc];
         for (i = h; i < n; i++) {
            val = arr[i];
            k   = ind[i];

            j = i;

            while ( (j >= h) && (arr[j-h] > val) ) {
               arr[j] = arr[j-h];
               ind[j] = ind[j-h];
               j = j - h;
            }

            arr[j] = val;
            ind[j] = k;
         }
      }
   }
   else { // sort in decreasing order
      for (; numinc >= 0; numinc--) {
         h = inc[numinc];
         for (i = h; i < n; i++) {
            val = arr[i];
            k   = ind[i];

            j = i;

            while ( (j >= h) && (arr[j-h] < val) ) {
               arr[j] = arr[j-h];
               ind[j] = ind[j-h];
               j = j - h;
            }

            arr[j] = val;
            ind[j] = k;
         }
      }
   }
}
